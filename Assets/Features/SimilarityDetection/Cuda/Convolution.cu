extern "C" {
	
#include <hip/hip_runtime.h>
#include <stdio.h>
	class Color {
	public:
		float r;
		float g;
		float b;
		float a;
	};
	// __device__ void multiply_int_color(int int_val, Color* color_val){
	// 	color_val->r = color_val->r * (float)int_val;
	// 	color_val->g = color_val->g * (float)int_val;
	// 	color_val->b = color_val->b * (float)int_val;
	// }
	__device__ void add_multiply_color_intcolor(Color* result, int int_val, Color* color_val){
		result->r+= color_val->r * ((float)int_val);
		result->g+= color_val->g * ((float)int_val);
		result->b+= color_val->b * ((float)int_val);
	}

	__device__ void div_color_int(Color* result, int int_val){
		result->r/=(float)int_val;
		result->g/=(float)int_val;
		result->b/=(float)int_val;
	}
	// Foreach pixel, perform convolution with filter, output is put back in input array
	__global__ void convolution(Color* pic1Pixels, 
								int columns, int rows,
								int* filter, int fColumns, int fRows,
								int lengthWithWeights) {
		int index = blockDim.x * blockIdx.x + threadIdx.x ; 
		 if(index < columns * rows){	
			Color accum = {0.0f, 0.0f, 0.0f, 0.0f};
			for(int fR = 0; fR < fRows; ++fR){
				for(int fC = 0; fC < fColumns; ++fC){
					// Convolution
					add_multiply_color_intcolor(&accum, filter[fR * fColumns + fC],
											    &pic1Pixels[index + fR*columns + fC ] );
				}
			}
			// Normalize differences
 			div_color_int(&accum, lengthWithWeights);
			pic1Pixels[index] = accum;
		}
	}

	// __global__ void buffer(Color* imgPixels, Color* bufImgPixels, 
	// 							int  columns, int rows, int pixel, Color color,
	// 							int imgWidth, int imgHeight){

	// 	int index = blockDim.x * blockIdx.x + threadIdx.x;
	// 	if(index < columns * rows){
	// 			int temp = threadIdx.x - blockDim.x * blockIdx.x;

	// 			int block_i = sqrt((float)blockDim.x);
	// 			int blockRow = (int)(threadIdx.x % block_i);
	// 			int blockColumn = (int)(threadIdx.x / block_i);

	// 			int grid_length = sqrt((float)gridDim.x);
	// 			int gridRow = (int)(blockIdx.x % grid_length);
	// 			int gridColumn = (int)(blockIdx.x / (float)grid_length);

	// 		    if ((gridRow == 0 ) &&true)//(blockRow == 0 || blockColumn ==  block_i-1))
    //             {
    //                 bufImgPixels[index] =  Color{0,0,1,1};
    //             }
    //             else if (blockRow >= imgHeight + pixel || blockColumn >= imgWidth + pixel)
    //             {
    //                 bufImgPixels[index] = Color{0, 1,0,1};
    //             }
    //             else {
    //                 //bufImgPixels[index] = imgPixels[index-pixel];
    //                 bufImgPixels[index] = Color{1, 0,0,1};

    //             }
	// 	}
	// }
	
	__device__ void color_abs(Color* color_val){
		color_val->r = abs(color_val->r);
		color_val->g = abs(color_val->g);
		color_val->b = abs(color_val->b);
		color_val->a = abs(color_val->a);
	}

	__device__ void getDistance(Color* color_val, float* output){
		*output = sqrt(color_val->r*color_val->r + color_val->g*color_val->g + color_val->b*color_val->b);
	}


	__device__ float lerp(const float a, const float b,  const float w, float* output)
	{
    	*output =  a + w*(b-a);
	}
	__device__ float lerp_vec(const Color* a,const Color* b,const float w, Color* output)
	{
		lerp(a->r, b->r, w, &output->r);
		lerp(a->g, b->g, w, &output->g);
		lerp(a->b, b->b, w, &output->b);
	}

	// foreach pixl, calculate pixelwise difference between the two arrays and output the modified difference in the first array
	__global__ void diff(Color* pic1Pixels, Color* pic2Pixels, 
								int  columns, int rows, float Threshold, float* diffAccumulator){
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if(index < columns * rows){
				//if pixel is in padded area, ignore it
				if(pic1Pixels[index].a == 1.0f){
                	pic1Pixels[index]=  Color{0.0f, 0.0f, 0.0f, 0.0f};
				}
				else{  
					// Absolute Component-wise difference
					Color diff = Color{0.0f, 0.0f, 0.0f, 1.0f};
					diff.r =  pic1Pixels[index].r - pic2Pixels[index].r;
					diff.g =  pic1Pixels[index].g - pic2Pixels[index].g;
					diff.b =  pic1Pixels[index].b - pic2Pixels[index].b;
					diff.a =  pic1Pixels[index].a - pic2Pixels[index].a;
					color_abs(&diff);
					// Distance of color from 0
                	float diffNumber=0;
					getDistance(&diff, &diffNumber);
					// If difference between threshold, ignore it
                	if (diffNumber < Threshold)
                	{
                	    diff = Color{0.0f, 0.0f, 0.0f, 0.0f};
						diffNumber = 0;
                	}
                	else {
                	    Color temp = diff;

						// From threshold to maxdistance, how far is our color distance?
                	    float t1 =0;

						lerp((float)Threshold, 1.732f, (float)diffNumber, &t1);
						Color diff = Color{0.0, 0.0, 0.0, 1.0};
						Color black = Color{0.0, 0.0, 0.0, 0.0};

						// Based on how far our distance is, interpolate our color from minimum to maximum
                	    lerp_vec(&black, &temp, t1, &diff);
                	}
                	pic1Pixels[index] = diff;
					// Add modified difference
					getDistance(&diff, &diffNumber);
                	*diffAccumulator += diffNumber;
				}
		}

	}

	//foreach pixel
	__global__ void overlay(Color* colors, int  columns, int rows, float Threshold){
		int index = blockDim.x * blockIdx.x + threadIdx.x;

		if(index < columns * rows){
			
                float remappedDistance = 0.0f;
				getDistance(&colors[index], &remappedDistance);
				// remap from [0.7, 1] to [0, 1]
                remappedDistance = (remappedDistance - Threshold) / (1.0f - Threshold);
                colors[index] = Color{1.0f, 0.0f, 0.0f, remappedDistance};
		}

	}

	
	int main(){}
}