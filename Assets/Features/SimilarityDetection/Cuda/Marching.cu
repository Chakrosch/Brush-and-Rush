extern "C" {
	
#include <hip/hip_runtime.h>
#include <stdio.h>
	class Vector3 {
	public:
		float x;
        float y;
        float z;
	};


	// foreach pixl, calculate pixelwise difference between the two arrays and output the modified difference in the first array
	__global__ void carve(int width, int height, int depth, int x, int y, int z, float* voxels){
        int new_x = x+threadIdx.x - 1;
        int new_y = y+threadIdx.y - 1;
        int new_z = z+threadIdx.z - 1;

		if( new_x < width  && new_x>0 &&  new_y < height  && new_y>0 &&  new_z < depth  && new_z>0){
            voxels[new_z * width * height + new_y * width + new_x] = -1;
		}

	}

	int main(){}
}