extern "C" {
	
#include <hip/hip_runtime.h>
#include <stdio.h>
    #include <stdlib.h>
	class Vector3 {
	public:
		float x;
        float y;
        float z;
	};

	__global__ void carve(int width, int height, int depth, int x, int y, int z, float* voxels){
        int new_x = x+threadIdx.x - 1;
        int new_y = y+threadIdx.y - 1;
        int new_z = z+threadIdx.z - 1;

		if( new_x < width  && new_x>0 &&  new_y < height  && new_y>0 &&  new_z < depth  && new_z>0){
            voxels[new_x * height * depth + new_y * depth + new_z] = -1;
		}

	}
    __device__ Vector3* normalize_inversed(Vector3* vec){
        float length = sqrt(pow(vec->x, 2) + pow(vec->y, 2) + pow(vec->z, 2));
        vec->x = (vec->x / length) * -1;
        vec->y = (vec->y / length) * -1; 
        vec->z = (vec->z / length) * -1;
        return vec;
    }
    __device__ float Lerp(float v0, float v1, float t)
    {
        return v0 + (v1 - v0) * t;
    }

     __device__ float BLerp(float v00, float v10, float v01, float v11, float tx, float ty)
    {
        return Lerp(Lerp(v00, v10, tx), Lerp(v01, v11, tx), ty);
    }

    __device__ float clamp(float d, float min, float max){
        const float t = d < min ? min : d;
        return t > max ? max : t;
    }

    __device__ float GetVoxel_int(int x, int y, int z, int Width, int Height, int Depth, float* Voxels){
        x = clamp(x, 0, Width - 1);
        y = clamp(y, 0, Height - 1);
        z = clamp(z, 0, Depth - 1);
        return Voxels[x * Height * Depth + y * Depth + z];
    }



    __device__ float GetVoxel(float u, float v, float w, int Width, int Height, int Depth, float* Voxels)
    {
        float x = u * (Width - 1);
        float y = v * (Height - 1);
        float z = w * (Depth - 1);

        int xi = (int)floor(x);
        int yi = (int)floor(y);
        int zi = (int)floor(z);

        float v000 = GetVoxel_int(xi    , yi    , zi    , Width, Height, Depth, Voxels);
        float v100 = GetVoxel_int(xi + 1, yi    , zi    , Width, Height, Depth, Voxels);
        float v010 = GetVoxel_int(xi    , yi + 1, zi    , Width, Height, Depth, Voxels);
        float v110 = GetVoxel_int(xi + 1, yi + 1, zi    , Width, Height, Depth, Voxels);

        float v001 = GetVoxel_int(xi    , yi    , zi + 1        , Width, Height, Depth, Voxels);
        float v101 = GetVoxel_int(xi + 1, yi    , zi + 1    , Width, Height, Depth, Voxels);
        float v011 = GetVoxel_int(xi    , yi + 1, zi + 1    , Width, Height, Depth, Voxels);
        float v111 = GetVoxel_int(xi + 1, yi + 1, zi + 1, Width, Height, Depth, Voxels);

        float tx = clamp(x - xi, 0 ,1);
        float ty = clamp(y - yi, 0 ,1);
        float tz = clamp(z - zi, 0, 1);

        //use bilinear interpolation the find these values.
        float v0 = BLerp(v000, v100, v010, v110, tx, ty);
        float v1 = BLerp(v001, v101, v011, v111, tx, ty);

        return Lerp(v0, v1, tz);
    }

    __global__ void GetNormal(int N, int Width, int Height, int Depth, Vector3* verts, float* Voxels, Vector3* output)
    {
		int index = blockDim.x * blockIdx.x + threadIdx.x; 
        if(index < N){
            float u = verts[index].x / (Width - 1.0f);
            float v = verts[index].y / (Height - 1.0f);
            float w = verts[index].z / (Depth - 1.0f);

            const float h = 0.005f;
            const float hh = h * 0.5f;
            const float ih = 1.0f / h;

            float dx_p1 = GetVoxel(u + hh, v     , w     , Width, Height, Depth, Voxels);
            float dy_p1 = GetVoxel(u     , v + hh, w     , Width, Height, Depth, Voxels);
            float dz_p1 = GetVoxel(u     , v     , w + hh, Width, Height, Depth, Voxels);

            float dx_m1 = GetVoxel(u - hh, v     , w     , Width, Height, Depth, Voxels);
            float dy_m1 = GetVoxel(u     , v - hh, w     , Width, Height, Depth, Voxels);
            float dz_m1 = GetVoxel(u     , v     , w - hh, Width, Height, Depth, Voxels);

            float dx = (dx_p1 - dx_m1) * ih;
            float dy = (dy_p1 - dy_m1) * ih;
            float dz = (dz_p1 - dz_m1) * ih;
            Vector3 out_temp =  Vector3{dx, dy, dz};
            output[index] = *normalize_inversed(&out_temp);
            //output[index] = Vector3{0.57, 0.57, 0.57};
        }
    }


	int main(){}
}